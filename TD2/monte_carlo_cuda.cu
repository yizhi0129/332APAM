#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "omp.h"    

#define N_BLOCKS 512
#define N_THREADS_PER_BLOCK 256
#define TRIALS_PER_THREAD 10E10

__global__ void monte_carlo(double* pi_d)
{
    // Variables
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int n_success = 0;
    float x, y;
    unsigned long long seed = 1234;
    hiprandState_t state;
    hiprand_init(seed, id, 0, &state);

    // Loop over trials
    if (id < N_BLOCKS * N_THREADS_PER_BLOCK) 
    {   
        for (int i = 0; i < TRIALS_PER_THREAD; i ++)
        {
            x = hiprand_uniform(&state);
            y = hiprand_uniform(&state);
            if (x * x + y * y < 1.0)
                n_success ++;
        }
        // Store pi values
        pi_d[id] = (double)n_success * 4.0 / TRIALS_PER_THREAD;
    }
}

int main(int argv, char** argc)
{
    // Variables
    double* pi, *pi_d;
    double avg_pi = 0.0;

    float start, stop;
    hipEvent_t start_event, stop_event;
    
    // Allocate memory
    pi = (double*)malloc(N_THREADS_PER_BLOCK * N_BLOCKS * sizeof(double));
    hipMalloc(&pi_d, N_THREADS_PER_BLOCK * N_BLOCKS * sizeof(double));
    
    // Create events
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    
    // Start timer
    hipEventRecord(start_event, 0);
    
    // Launch kernel
    dim3  dimBlock(N_BLOCKS, 1, 1);
    dim3  dimGrid((N_THREADS_PER_BLOCK + dimBlock.x - 1)/dimBlock.x, 1, 1);
    monte_carlo<<<dimGrid, dimBlock>>>(pi_d);
    
    // Copy success to host
    hipMemcpy(pi, pi_d, N_THREADS_PER_BLOCK * sizeof(int), hipMemcpyDeviceToHost);
    
    // Stop timer
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&start, start_event, stop_event);
    hipEventElapsedTime(&stop, stop_event, stop_event);
    
    // Compute average pi parallel
    void compute_avg_pi(float *pi) 
    {
        float avg_pi = 0.0;
        #pragma omp parallel for reduction(+:avg_pi)
        for (int i = 0; i < N_THREADS_PER_BLOCK * N_BLOCKS; i++) 
        {
            avg_pi += pi[i];
        }
        avg_pi /= N_THREADS_PER_BLOCK * N_BLOCKS;
        printf("Average pi: %f\n", avg_pi);
    }

    // Free memory   
    hipFree(pi_d);
    free(pi);

    // Print results
    printf("Pi: %f\n", avg_pi);
    printf("Time: %f ms\n", stop - start);

    return 0;
}
