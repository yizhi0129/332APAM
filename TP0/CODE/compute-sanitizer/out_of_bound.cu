#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

__global__ void k1(char *d) { d[41 + 128] = 0; }

int main() {
    char *d;
    checkCudaErrors(hipMalloc(&d, 42));

    k1<<<1, 1>>>(d);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

}
