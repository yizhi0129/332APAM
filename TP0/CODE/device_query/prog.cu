#include "hip/hip_runtime.h"
//
//  Created by Patricio Bulic, Davor Sluga, UL FRI on 6/6/2022.
//  Copyright © 2022 Patricio Bulic, Davor Sluga UL FRI. All rights reserved.
//

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "hip/hip_runtime_api.h"

int main(int argc, char **argv) 
{
  
       // Get number of GPUs
       int deviceCount = 0;
       hipError_t error = hipGetDeviceCount(&deviceCount);

       if (error != hipSuccess) 
       {
       printf("hipGetDeviceCount error %d\n-> %s\n", error, hipGetErrorString(error));
       exit(EXIT_FAILURE);
       }

       // Get device propreties and print 
       for (int dev = 0; dev < deviceCount; dev++) 
       {
       struct hipDeviceProp_t prop;
       int value, mem_clock_rate, mem_bus_width, GPU_cores, GPU_clock_rate;
       printf("\n==========  cudaDeviceGetProperties ============  \n");
       hipGetDeviceProperties(&prop, dev);
       printf("\nDevice %d: \"%s\"\n", dev, prop.name);
       GPU_clock_rate = prop.clockRate; // kHz
       printf("  GPU Clock Rate (MHz):                          %d\n", 
              GPU_clock_rate / 1000);
       printf("  Memory Clock Rate (MHz):                       %d\n", 
              prop.memoryClockRate/1000);
       printf("  Memory Bus Width (bits):                       %d\n", 
              prop.memoryBusWidth);
       printf("  CUDA Cores/MP:                                 %d\n",
              _ConvertSMVer2Cores(prop.major, prop.minor));
       GPU_cores = _ConvertSMVer2Cores(prop.major, prop.minor) * prop.multiProcessorCount;
       printf("  CUDA Cores:                                    %d\n", 
              GPU_cores);
       printf("  Total amount of global memory:                 %.0f GB\n", 
              prop.totalGlobalMem / 1073741824.0f);
       printf("  Total amount of shared memory per block:       %zu kB\n",
              prop.sharedMemPerBlock/1024);
       printf("  Total number of registers available per block: %d\n",
              prop.regsPerBlock);
       printf("  Warp size:                                     %d\n",
              prop.warpSize);
       printf("  Maximum number of threads per block:           %d\n",
              prop.maxThreadsPerBlock);
       printf("  Max dimension size of a thread block (x,y,z): (%d, %d, %d)\n",
              prop.maxThreadsDim[0], prop.maxThreadsDim[1],
              prop.maxThreadsDim[2]);
       printf("  Max dimension size of a grid size    (x,y,z): (%d, %d, %d)\n",
              prop.maxGridSize[0], prop.maxGridSize[1],
              prop.maxGridSize[2]);
       printf("  Peak FP32 operations per second in GFLOPS:    %d\n", 
              2 * GPU_cores * GPU_clock_rate / 1e6); 
              // use GPU cores, GPU clock rate (kHz) to calculate peak FP32 operations per second in GFLOPS

       printf("\n\n==========  hipDeviceGetAttribute ============  \n");
       printf("\nDevice %d: \"%s\"\n", dev, prop.name);
       hipDeviceGetAttribute (&value, hipDeviceAttributeMaxThreadsPerBlock, dev);
       printf("  Max number of threads per block:              %d\n",
              value);
       hipDeviceGetAttribute (&value, hipDeviceAttributeMaxBlockDimX, dev);
       printf("  Max block dimension X:                        %d\n",
              value);
       hipDeviceGetAttribute (&value, hipDeviceAttributeMaxBlockDimY, dev);
       printf("  Max block dimension Y:                        %d\n",
              value);
       hipDeviceGetAttribute (&value, hipDeviceAttributeMaxBlockDimZ, dev);
       printf("  Max block dimension Z:                        %d\n",
              value);
       hipDeviceGetAttribute (&value, hipDeviceAttributeMaxGridDimX, dev);
       printf("  Max grid dimension X:                         %d\n",
              value);
       hipDeviceGetAttribute (&value, hipDeviceAttributeMaxGridDimY, dev);
       printf("  Max grid dimension Y:                         %d\n",
              value);
       hipDeviceGetAttribute (&value, hipDeviceAttributeMaxGridDimZ, dev);
       printf("  Max grid dimension Z:                         %d\n",
              value);
       hipDeviceGetAttribute (&value, hipDeviceAttributeMaxSharedMemoryPerBlock, dev);
       printf("  Max shared memory per block:                  %d\n",
              value);
       hipDeviceGetAttribute (&value, hipDeviceAttributeWarpSize, dev);
       printf("  Warp size:                                    %d\n",
              value);      
       hipDeviceGetAttribute (&value, hipDeviceAttributeClockRate, dev);
       printf("  Peak clock frequency in kilohertz:            %d\n",
              value);
       hipDeviceGetAttribute (&mem_clock_rate, hipDeviceAttributeMemoryClockRate, dev);
       printf("  Peak memory clock frequency in kilohertz:     %d\n",
              mem_clock_rate);
       hipDeviceGetAttribute (&mem_bus_width, hipDeviceAttributeMemoryBusWidth, dev);
       printf("  Global memory bus width in bits:              %d\n",
              mem_bus_width);
       hipDeviceGetAttribute (&value, hipDeviceAttributeL2CacheSize, dev);
       printf("  Size of L2 cache in bytes:                    %d\n",
              value);
       hipDeviceGetAttribute (&value, hipDeviceAttributeMaxThreadsPerMultiProcessor, dev);
       printf("  Maximum resident threads per SM:              %d\n",
              value);
       hipDeviceGetAttribute (&value, hipDeviceAttributeComputeCapabilityMajor, dev);
       printf("  Major compute capability version number:      %d\n",
              value);
       hipDeviceGetAttribute (&value, hipDeviceAttributeComputeCapabilityMinor, dev);
       printf("  Minor compute capability version number:      %d\n",
              value);
       hipDeviceGetAttribute (&value, hipDeviceAttributeMaxSharedMemoryPerMultiprocessor, dev);
       printf("  Max shared memory per SM in bytes:            %d\n",
              value);
       hipDeviceGetAttribute (&value, hipDeviceAttributeMaxRegistersPerMultiprocessor, dev);
       printf("  Max number of 32-bit registers per SM:        %d\n",
              value);  
       hipDeviceGetAttribute (&value, hipDeviceAttributeSharedMemPerBlockOptin, dev);
       printf("  Max per block shared mem size on the device:  %d\n",
              value);  
       hipDeviceGetAttribute (&value, hipDeviceAttributeMaxBlocksPerMultiprocessor, dev);
       printf("  Max thread blocks that can reside on a SM:    %d\n",
              value);  
       printf("  Peak memory bandwidth in GB/s:                %.2f\n",
              2.0 * mem_clock_rate * (mem_bus_width / 8) / 1.0e6); 
              // use Memory Clock Rate (kHz) and Memory Bus Width (bit) to calculate memory bandwidth (GB/s)       
  }
}
